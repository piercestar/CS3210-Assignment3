/**
 * 
 * Matrix Multiplication - CUDA for GPUs
 *
 * CS3210
 *
 **/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

int size;
#define BLOCK_SIZE 32

typedef struct
{
	float ** element;
} matrix;


long long wall_clock_time()
{
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

/**
 * Allocates memory for a matrix of size SIZE
 * The memory is allocated row-major order, i.e. 
 *  elements from the same row are allocated at contiguous 
 *  memory addresses.
 **/
void allocate_matrix(matrix* m)
{
	int i;
	hipError_t rc;
	
	// allocate array for all the rows
	rc = hipMallocManaged((void**)&(m->element), sizeof(float*) * size);
	if (rc != hipSuccess)
	{
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
		exit(1);
	}
	
	// allocate an array for each row of the matrix
	for (i = 0; i < size; i++)
	{
		rc = hipMallocManaged((void**)&(m->element[i]), sizeof(float) * size);
		if (rc != hipSuccess)
		{
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
			exit(1);
		}
	}
}

/**
 * Free the memory allocated for a matrix.
 **/
void free_matrix(matrix* m) {
	int i;
	for (i = 0; i < size; i++)
		hipFree(m->element[i]);
	hipFree(m->element);
}

/**
 * Initializes the elements of the matrix with
 * random values between 0 and 9
 **/
void init_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = rand() % 10;
		}
}

/**
 * Initializes the elements of the matrix with
 * element 0.
 **/
void init_matrix_zero(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = 0.0;
		}
}


/**
 * Multiplies matrix @a with matrix @b storing
 * the result in matrix @result
 * 
 * The multiplication algorithm is the O(n^3) 
 * algorithm
 */
void mm(matrix a, matrix b, matrix result)
{
	int i, j, k;
	
	// Do the multiplication
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			for(k = 0; k < size; k++)
				result.element[i][j] += a.element[i][k] * b.element[k][j];
}

__device__ void print_sm(float a[BLOCK_SIZE][BLOCK_SIZE]) 
{
	int i,j;
	int block = 1;
	if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == block && blockIdx.y == block) {
		printf("printing sm...\n");
		for (i = 0; i < BLOCK_SIZE; i++) {
			printf("row %d :", i);
			for(j = 0; j < BLOCK_SIZE; j++) 
				printf("%1.2f ", a[i][j]);
			printf("\n");
		}
	}
}

/**
 * Each kernel computes the result element (i,j).
 */
__global__ void mm_kernel(matrix a, matrix b, matrix result, int size)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x; 
	int k,e;

	if (i >= size || j >= size)
		return;

	int Cvalue = 0;
	
	for(k = 0; k < size/BLOCK_SIZE; k++) {
	
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[threadIdx.y][threadIdx.x] = a.element[blockIdx.y * blockDim.y + threadIdx.y][k * BLOCK_SIZE + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = b.element[k * BLOCK_SIZE + threadIdx.y][blockIdx.x * blockDim.x + threadIdx.x];	
		
		__syncthreads();
	
		for (e = 0; e < BLOCK_SIZE; e++) {
			Cvalue += As[threadIdx.y][e] * Bs[e][threadIdx.x];
		}
		
		__syncthreads();

	}
	result.element[i][j] = Cvalue;
}

void print_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
	{
		printf("row %4d: ", i);
		for (j = 0; j < size; j++)
			printf("%6.2f  ", m.element[i][j]);
		printf("\n");
	}
}



void work()
{
	matrix a, b, result1, result2;
	long long before, after;
	int correct, i, j, dim;
	hipError_t rc;

	// Allocate memory for matrices
	allocate_matrix(&a);
	allocate_matrix(&b);
	allocate_matrix(&result1);
	allocate_matrix(&result2);	

	// Initialize matrix elements
	init_matrix(a);
	init_matrix(b);
	
	// Perform sequential matrix multiplication
	before = wall_clock_time();
	mm(a, b, result1);
	after = wall_clock_time();
        fprintf(stderr, "Matrix multiplication on CPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// Perform CUDA matrix  multiplication
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);			// a block of BLOCK_SIZE CUDA threads
	dim = (size % BLOCK_SIZE == 0) ? size / BLOCK_SIZE : size / BLOCK_SIZE + 1; 
	dim3 grid(dim, dim);	// a grid of CUDA thread blocks
	before = wall_clock_time();
	mm_kernel<<<grid, block>>>(a, b, result2, size);
	hipDeviceSynchronize();
	after = wall_clock_time();
	fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

	// Compare the results
	correct = 1;
	for (i = 0; correct && i < size; i++)
		for (j = 0; j < size; j++)
			if (result1.element[i][j] != result2.element[i][j]) {
				correct = 0;
				break;
			}

	if (correct) {
		printf("The result matrices are identical!\n");
	
	} else {
		printf("Difference in result matrices at element (%d, %d)!\n", i, j);
		//print_matrix(result1);
		//print_matrix(result2);
	}

	free_matrix(&a);
	free_matrix(&b);
	free_matrix(&result1);
	free_matrix(&result2);
}


int main(int argc, char ** argv)
{
	srand(0); 

	printf("Usage: %s <size>\n", argv[0]);
    
	if (argc >= 2)
		size = atoi(argv[1]);
	else
		size = 1024;
		
	fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
	// Multiply the matrices
	work();

	return 0;
}
